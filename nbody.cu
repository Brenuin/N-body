#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <fstream>
#include <iomanip>
#include <omp.h>
#include <hip/hip_runtime.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef NUMPLANETS
#define NUMPLANETS 4000
#endif

#ifndef NUMSTARS
#define NUMSTARS 500
#endif

#define G 6.67430e-11

struct PointMass {
    double mass;
    double position[3];
    double velocity[3];
    double force[3];
    double angularVelocity[3];
    double torque[3];
    double momentOfInertia;
};

void displayLoadingBar(int step, int steps) {
    int barWidth = 70;
    float progress = (float)step / steps;

    std::cout << "[";
    int pos = barWidth * progress;
    for (int i = 0; i < barWidth; ++i) {
        if (i < pos) std::cout << "=";
        else if (i == pos) std::cout << ">";
        else std::cout << " ";
    }
    std::cout << "] " << int(progress * 100.0) << " %\r";
    std::cout.flush();
}

__device__ double computeDistance(const PointMass &a, const PointMass &b) {
    double dx = b.position[0] - a.position[0];
    double dy = b.position[1] - a.position[1];
    double dz = b.position[2] - a.position[2];
    return sqrt(dx * dx + dy * dy + dz * dz);
}

__device__ void computeForce(const PointMass &a, const PointMass &b, double *force) {
    double distance = computeDistance(a, b);
    if (distance == 0) return;

    double magnitude = G * a.mass * b.mass / (distance * distance);

    force[0] = magnitude * (b.position[0] - a.position[0]) / distance;
    force[1] = magnitude * (b.position[1] - a.position[1]) / distance;
    force[2] = magnitude * (b.position[2] - a.position[2]) / distance;
}

__device__ void computeTorque(const PointMass &a, const double *force, double *torque) {
    torque[0] = a.position[1] * force[2] - a.position[2] * force[1];
    torque[1] = a.position[2] * force[0] - a.position[0] * force[2];
    torque[2] = a.position[0] * force[1] - a.position[1] * force[0];
}

__global__ void updateForcesAndTorques(PointMass *masses, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    masses[i].force[0] = masses[i].force[1] = masses[i].force[2] = 0.0;
    masses[i].torque[0] = masses[i].torque[1] = masses[i].torque[2] = 0.0;

    for (int j = 0; j < n; ++j) {
        if (i != j) {
            double force[3];
            computeForce(masses[i], masses[j], force);
            masses[i].force[0] += force[0];
            masses[i].force[1] += force[1];
            masses[i].force[2] += force[2];

            double torque[3];
            computeTorque(masses[i], force, torque);
            masses[i].torque[0] += torque[0];
            masses[i].torque[1] += torque[1];
            masses[i].torque[2] += torque[2];
        }
    }
}

__global__ void updateKinematics(PointMass *masses, int n, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    masses[i].velocity[0] += masses[i].force[0] / masses[i].mass * dt;
    masses[i].velocity[1] += masses[i].force[1] / masses[i].mass * dt;
    masses[i].velocity[2] += masses[i].force[2] / masses[i].mass * dt;

    masses[i].position[0] += masses[i].velocity[0] * dt;
    masses[i].position[1] += masses[i].velocity[1] * dt;
    masses[i].position[2] += masses[i].velocity[2] * dt;

    masses[i].angularVelocity[0] += masses[i].torque[0] / masses[i].momentOfInertia * dt;
    masses[i].angularVelocity[1] += masses[i].torque[1] / masses[i].momentOfInertia * dt;
    masses[i].angularVelocity[2] += masses[i].torque[2] / masses[i].momentOfInertia * dt;
}

void generate_galaxy(std::vector<PointMass>& masses, int numPlanets, int numStars, const double* center, double radius, double mass_min, double mass_max, const double* initialVelocity = nullptr) {
    int startIdx = masses.size();
    int totalBodies = numPlanets + numStars;
    double totalMass = 0.0;
    double mass;

    auto randomInRange = [](double min, double max) {
        return min + static_cast<double>(rand()) / RAND_MAX * (max - min);
    };

    // Generate planets
    for (int i = 0; i < numPlanets; ++i) {
        double position[3];
        double distance = randomInRange(0, radius);
        double angle = randomInRange(0, 2 * M_PI);
        position[0] = center[0] + distance * cos(angle);
        position[1] = center[1] + distance * sin(angle);
        position[2] = center[2] + randomInRange(-radius, radius);

        double velocity[3] = {0.0, 0.0, 0.0};
        double force[3] = {0.0, 0.0, 0.0};
        double angularVelocity[3] = {0.0, 0.0, 0.0};
        double torque[3] = {0.0, 0.0, 0.0};
        double momentOfInertia = randomInRange(1e10, 1e20);

        mass = randomInRange(100 * mass_min, 100 * mass_max);
        totalMass += mass;

        masses.push_back(PointMass{mass, {position[0], position[1], position[2]}, {velocity[0], velocity[1], velocity[2]}, {force[0], force[1], force[2]}, {angularVelocity[0], angularVelocity[1], angularVelocity[2]}, {torque[0], torque[1], torque[2]}, momentOfInertia});
    }

    // generate stars
    for (int i = 0; i < numStars; ++i) {
        double position[3];
        double distance = randomInRange(0, radius);
        double angle = randomInRange(0, 2 * M_PI);
        position[0] = center[0] + distance * cos(angle);
        position[1] = center[1] + distance * sin(angle);
        position[2] = center[2] + randomInRange(-radius, radius);

        double velocity[3] = {0.0, 0.0, 0.0};
        double force[3] = {0.0, 0.0, 0.0};
        double angularVelocity[3] = {0.0, 0.0, 0.0};
        double torque[3] = {0.0, 0.0, 0.0};
        double momentOfInertia = randomInRange(1e10, 1e20);

        mass = randomInRange(mass_min, mass_max);
        totalMass += mass;

        masses.push_back(PointMass{mass, {position[0], position[1], position[2]}, {velocity[0], velocity[1], velocity[2]}, {force[0], force[1], force[2]}, {angularVelocity[0], angularVelocity[1]}, {torque[0], torque[1], torque[2]}, momentOfInertia});
    }

    double averageMass = totalMass / totalBodies;

    // set stable orbital velocitys
    for (int i = startIdx; i < startIdx + totalBodies; ++i) {
        auto& mass = masses[i];
        double dx = mass.position[0] - center[0];
        double dy = mass.position[1] - center[1];
        double dz = mass.position[2] - center[2];
        double distance = sqrt(dx * dx + dy * dy + dz * dz);

        double orbitalVelocity = sqrt(G * averageMass / distance);

        double angle = randomInRange(0, 2 * M_PI);
        double sinAngle = sin(angle);
        double cosAngle = cos(angle);

        if (dx != 0 || dy != 0) {
            mass.velocity[0] = -dy / distance * orbitalVelocity * cosAngle;
            mass.velocity[1] = dx / distance * orbitalVelocity * sinAngle;
            mass.velocity[2] = 0.0;
        } else {
            mass.velocity[0] = orbitalVelocity * cosAngle;
            mass.velocity[1] = orbitalVelocity * sinAngle;
            mass.velocity[2] = 0.0;
        }

        double normalComponent = (mass.velocity[0] * dx + mass.velocity[1] * dy) / (dx * dx + dy * dy);
        mass.velocity[0] -= normalComponent * dx;
        mass.velocity[1] -= normalComponent * dy;
        mass.velocity[2] = orbitalVelocity * sinAngle;

        // Add initial velocity if provided
        if (initialVelocity) {
            mass.velocity[0] += initialVelocity[0];
            mass.velocity[1] += initialVelocity[1];
            mass.velocity[2] += initialVelocity[2];
        }
    }
}


int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    srand(static_cast<unsigned int>(time(NULL)));

    double randsize_min = 1.0e23, randsize_max = 1.0e30;
    double galaxyRadius = 24.0e14;

    std::vector<PointMass> masses;
    double galaxyCenter1[3] = {0.0, 0.0, 0.0};
    double galaxyCenter2[3] = {70.0e14, 0.0, 0.0};

    double initialVelocity1[3] = {90000.0, 0.0, 0.0};
    double initialVelocity2[3] = {-80000.0, 0.0, 0.0};

    generate_galaxy(masses, NUMPLANETS, NUMSTARS, galaxyCenter1, galaxyRadius, randsize_min, randsize_max, initialVelocity1);
    generate_galaxy(masses, NUMPLANETS, NUMSTARS, galaxyCenter2, galaxyRadius, randsize_min, randsize_max, initialVelocity2);


    PointMass *d_masses;
    hipMalloc(&d_masses, masses.size() * sizeof(PointMass));
    hipMemcpy(d_masses, masses.data(), masses.size() * sizeof(PointMass), hipMemcpyHostToDevice);

    double dt = 7000000;//.0001;//5;//100;//10000;//
    int steps = 100;
    int numThreads = 256;
    int numBlocks = (masses.size() + numThreads - 1) / numThreads;

    // open the output file once
    std::ofstream outFile("simulation_data.txt");

    // start timing
    hipEventRecord(start);

    for (int step = 0; step < steps; ++step) {
        updateForcesAndTorques<<<numBlocks, numThreads>>>(d_masses, masses.size());
        updateKinematics<<<numBlocks, numThreads>>>(d_masses, masses.size(), dt);
        
        // Copy data back to host to write to file
        hipMemcpy(masses.data(), d_masses, masses.size() * sizeof(PointMass), hipMemcpyDeviceToHost);

        // Write the positions of all masses at the current time step
        for (const auto& mass : masses) {
            outFile << mass.position[0] << " " << mass.position[1] << " " << mass.position[2] << " " << mass.mass << " ";
        }
        outFile << "\n"; 
        
        // Display loading bar
        displayLoadingBar(step, steps);

        // Debugging: Print step number
        std::cout << "Completed step " << step + 1 << " of " << steps << std::endl;
    }

    // Close the output file
    outFile.close();

   
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipFree(d_masses);

    double megaTrialsPerSecond = (double)(NUMPLANETS * NUMPLANETS * steps) / (milliseconds / 1000.0) / 1000000.0;
    fprintf(stderr, "NUMPLANETS: %8d, Performance: %6.2lf MegaTrials/Second\n", NUMPLANETS, megaTrialsPerSecond);

    std::ofstream perfFile("performance.csv", std::ios_base::app);
    if (perfFile.is_open()) {
        perfFile << NUMPLANETS << "," << megaTrialsPerSecond << "\n";
        perfFile.close();
    } else {
        std::cerr << "Unable to open performance.csv for writing\n";
    }

    // Complete the loading bar
    displayLoadingBar(steps, steps);
    std::cout << std::endl;

    return 0;
}
